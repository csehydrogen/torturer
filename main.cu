#include "hip/hip_runtime.h"
#include <cstdio>

#define CheckCUDA(exp) \
  do {\
    hipError_t status = (exp);\
    if (status != hipSuccess) {\
      fprintf(stderr, "[%s:%d] CUDA error: %s (code=%d)\n", \
          __FILE__, __LINE__, hipGetErrorString(status), static_cast<int>(status));\
      exit(EXIT_FAILURE);\
    }\
  } while (0)

const int NUM_GPU = 4;
const int BLOCK_SIZE = 64; // warp size * 2
const int BLOCK_NUM = 80 * 32; // (# of sm) * (max warp per sm / 2)
const size_t INPUT_NUM = 50790L; // (Input buffer size you want) / (BLOCK_NUM * BLOCK_SIZE * sizeof(float))

__global__ void kernel(float *in, float *out, int N, float fi0, float fi1) {
  int gid = blockIdx.x * blockDim.x + threadIdx.x;
  float fo0 = 0, fo1 = 0;
  for (int n = 0; n < N; n += 1) {
    float fi = in[(size_t)(n % INPUT_NUM) * BLOCK_NUM * BLOCK_SIZE + gid];
    fi0 += fi;
    fi1 += fi;
    for (int c = 0; c < 11; ++c) { // memory-to-compute ratio magic number
      fo0 += fi0;
      fo1 += fi1;
    }
  }
  out[gid] = fo0 * fo1;
}

double get_time() {
  struct timespec t;
  clock_gettime(CLOCK_MONOTONIC, &t);
  long x = 1000000000L * t.tv_sec + t.tv_nsec;
  return (double)x / (double)1000000000L;
}

int main() {

  dim3 blockDim(BLOCK_SIZE, 1, 1);
  dim3 gridDim(BLOCK_NUM, 1, 1);
  #pragma omp parallel for
  for (int j = 0; j < NUM_GPU; ++j) {
    hipSetDevice(j);
    float *in, *out;
    CheckCUDA(hipMalloc(&in, INPUT_NUM * BLOCK_NUM * BLOCK_SIZE * sizeof(float)));
    CheckCUDA(hipMalloc(&out, BLOCK_NUM * BLOCK_SIZE * sizeof(float)));
    while (true) {
      double st = get_time();
      kernel<<<gridDim, blockDim>>>(in, out, 1000000, 1.0, 2.0);
      hipDeviceSynchronize();
      double et = get_time();
      printf("[Device %d] %f s\n", j, et - st);
    }
  }

  return 0;
}
